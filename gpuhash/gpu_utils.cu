/*
* Copyright 2017-2018 Roman Klassen
*
* Licensed under the Apache License, Version 2.0 (the "License"); you may not
* use this file except in compliance with the License. You may obtain a copy
* of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
* WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
* License for the specific language governing permissions and limitations
* under the License.
*
*/

#include "gpu_utils.cuh"
#include "hip/hip_runtime.h"

void* gc_malloc(size_t bufsize)
{
	void* gpu_buf = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&gpu_buf, bufsize));
	return gpu_buf;
}

void gc_free(void* gpu_buf)
{
	CUDA_SAFE_CALL(hipFree(gpu_buf));
}

void* gc_host2device(gcStream_t Stream, void* cpu_buf, size_t bufsize)
{
	void* gpu_buf = NULL;

	unsigned int round_bufsize = CEIL(bufsize, 16) * 16 + 4;

	CUDA_SAFE_CALL(hipMalloc((void**)&gpu_buf, round_bufsize));
	CUDA_SAFE_CALL(hipMemcpy(gpu_buf, cpu_buf, bufsize, hipMemcpyHostToDevice));

	return gpu_buf;
}

void* gc_device2host(gcStream_t Stream, void* gpu_buf, size_t bufsize)
{
	void* pinned = NULL;
	CUDA_SAFE_CALL(hipHostMalloc((void**)&pinned, bufsize));
	CUDA_SAFE_CALL(hipMemcpyAsync(pinned, gpu_buf, bufsize, hipMemcpyDeviceToHost, (ihipStream_t*)Stream.stream));

	return pinned;
}

int align_size(int size, int align_by)
{
	int rest = size%align_by;
	if (rest == 0) return size;
	return size + rest;
}


void gc_stream_start(gcStream_t* Stream)
{
	CUDA_SAFE_CALL(hipStreamCreate((hipStream_t*)&Stream->stream));
	CUDA_SAFE_CALL(hipEventCreate((hipEvent_t*)&Stream->event));
	CUDA_SAFE_CALL(hipEventCreate((hipEvent_t*)&Stream->start));
	CUDA_SAFE_CALL(hipEventRecord((hipEvent_t)Stream->start, (hipStream_t)Stream->stream));
}

void gc_stream_stop(gcStream_t* Stream)
{
	CUDA_SAFE_CALL(hipEventRecord((hipEvent_t)Stream->event, (hipStream_t)Stream->stream));
	CUDA_SAFE_CALL(hipEventSynchronize((hipEvent_t)Stream->event));

	float etime = 0.0f;
	hipEventElapsedTime(&etime, (hipEvent_t)Stream->start, (hipEvent_t)Stream->event);
	printf("***%f ms\n", etime);
	CUDA_SAFE_CALL(hipEventDestroy((hipEvent_t)Stream->event));
	CUDA_SAFE_CALL(hipEventDestroy((hipEvent_t)Stream->start));
	CUDA_SAFE_CALL(hipStreamDestroy((hipStream_t)Stream->stream));
}

void gc_stream_wait(gcStream_t* Stream)
{
	CUDA_SAFE_CALL(hipStreamSynchronize((hipStream_t)Stream->stream));
}