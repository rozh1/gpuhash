#include "gpu_utils.cuh"
#include "hip/hip_runtime.h"


//=============================================================================
//Memory Management
//=============================================================================
void* gc_malloc(size_t bufsize)
{
	void* gpu_buf = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&gpu_buf, bufsize));
	return gpu_buf;
}

void gc_free(void* gpu_buf)
{
	CUDA_SAFE_CALL(hipFree(gpu_buf));
}

void* gc_host2device(gcStream_t Stream, void* cpu_buf, size_t bufsize)
{
	void* gpu_buf = NULL;

	unsigned int round_bufsize = CEIL(bufsize, 16) * 16 + 4;

	CUDA_SAFE_CALL(hipMalloc((void**)&gpu_buf, round_bufsize));
	CUDA_SAFE_CALL(hipMemcpy(gpu_buf, cpu_buf, bufsize, hipMemcpyHostToDevice));

	return gpu_buf;
}

void* gc_device2host(gcStream_t Stream, void* gpu_buf, size_t bufsize)
{
	void* pinned = NULL;
	CUDA_SAFE_CALL(hipHostMalloc((void**)&pinned, bufsize));
	CUDA_SAFE_CALL(hipMemcpyAsync(pinned, gpu_buf, bufsize, hipMemcpyDeviceToHost, (ihipStream_t*)Stream.stream));
	//void* cpu_buf = malloc(bufsize);
	//memcpy(cpu_buf, pinned, bufsize);
	//CUDA_SAFE_CALL(hipHostFree(pinned));

	//return cpu_buf;
	return pinned;
}

int align_size(int size, int align_by)
{
	int rest = size%align_by;
	if (rest == 0) return size;
	return size + rest;
}


void gc_stream_start(gcStream_t* Stream)
{
	CUDA_SAFE_CALL(hipStreamCreate((hipStream_t*)&Stream->stream));
	CUDA_SAFE_CALL(hipEventCreate((hipEvent_t*)&Stream->event));
	CUDA_SAFE_CALL(hipEventCreate((hipEvent_t*)&Stream->start));
	CUDA_SAFE_CALL(hipEventRecord((hipEvent_t)Stream->start, (hipStream_t)Stream->stream));
}

void gc_stream_stop(gcStream_t* Stream)
{
	CUDA_SAFE_CALL(hipEventRecord((hipEvent_t)Stream->event, (hipStream_t)Stream->stream));
	CUDA_SAFE_CALL(hipEventSynchronize((hipEvent_t)Stream->event));

	float etime = 0.0f;
	hipEventElapsedTime(&etime, (hipEvent_t)Stream->start, (hipEvent_t)Stream->event);
	printf("***%f ms\n", etime);
	CUDA_SAFE_CALL(hipEventDestroy((hipEvent_t)Stream->event));
	CUDA_SAFE_CALL(hipEventDestroy((hipEvent_t)Stream->start));
	CUDA_SAFE_CALL(hipStreamDestroy((hipStream_t)Stream->stream));
}

void gc_stream_wait(gcStream_t* Stream)
{
	CUDA_SAFE_CALL(hipStreamSynchronize((hipStream_t)Stream->stream));
}